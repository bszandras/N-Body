#include "hip/hip_runtime.h"
#define GLEW_STATIC

#include <Windows.h>

#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <GL/GLUtils.hpp>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>

#include <SDL.h>
#include <SDL_opengl.h>

#include <imgui/imgui.h>
#include <imgui/imgui_impl_sdl_gl3.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <cmath>

#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <corecrt_math_defines.h>

#define SOFTENING 1e-9f
#define BLOCK_SIZE 128

struct Particle {
    float x, y, z;
    float vx, vy, vz;
    float mass;
};

int lastMouseX = 0;
int lastMouseY = 0;
bool mousePressed = false;
glm::mat4 view;
float mouseAngleX = 0.0f;
float mouseAngleY = 0.8f;
float rotationSpeed = 0.01f;
float cameraDistance = 2;

glm::vec3 camUp;
glm::vec3 camForward;
glm::vec3 camRight;

bool uniform = false;
bool initialVel = true;
float gravSetting = 1;
float grav = 1;
int numParticles = BLOCK_SIZE * 80;
float startingParticleNum = numParticles;
int numIterations = 20000;
float dt = 0.0003f;
Particle* particlesHost;

void RotateCamera(int dx, int dy)
{
    mouseAngleX += dx * rotationSpeed;
    mouseAngleY += dy * rotationSpeed;
    if (mouseAngleY > 2.8)
    {
        mouseAngleY = 2.8;
    }
    else if (mouseAngleY < 0.2)
    {
        mouseAngleY = 0.2;
    }

    glm::vec3 at(0, 0, 0);
    glm::vec3 eye = at;
    eye.x += cameraDistance;
    glm::vec3 up(0, 0, 1);

    camForward = glm::normalize(at - eye);
    camRight = glm::normalize(glm::cross(camForward, up));
    camUp = glm::normalize(glm::cross(camRight, camForward));

    eye = at + cameraDistance * glm::vec3(cosf(mouseAngleX) * sinf(mouseAngleY),
        sinf(mouseAngleX) * sinf(mouseAngleY),
        cosf(mouseAngleY));

    view = glm::lookAt(eye, at, up);
}

// szimul�ci� ami CUDA
__global__ void nBodySimulation(Particle* currentParticles, Particle* lastParticles, float dt, int numParticles, float gravity) {
    // Get the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numParticles) {
        float ax = 0.0f, ay = 0.0f, az = 0.0f;

        float cx = currentParticles[tid].x;
        float cy = currentParticles[tid].y;
        float cz = currentParticles[tid].z;

        // Compute acceleration
        for (int j = 0; j < numParticles; j++) {
            float dx = lastParticles[j].x - cx;
            float dy = lastParticles[j].y - cy;
            float dz = lastParticles[j].z - cz;
            float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
            //float invDist = rsqrtf(distSqr);
            float invDist = sqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;

            float force = lastParticles[j].mass * gravity * invDist3;
            //float force = 1 * invDist3;

            ax += dx * force;
            ay += dy * force;
            az += dz * force;
        }

        // Update velocity
        currentParticles[tid].vx += ax * dt;
        currentParticles[tid].vy += ay * dt;
        currentParticles[tid].vz += az * dt;

        // Update position
        currentParticles[tid].x += currentParticles[tid].vx * dt;
        currentParticles[tid].y += currentParticles[tid].vy * dt;
        currentParticles[tid].z += currentParticles[tid].vz * dt;
    }
}

// sdl inicializ�ci�
SDL_Window* win;
SDL_GLContext context;
SDL_Event event;
void InitSDLWindow()
{
    if (SDL_Init(SDL_INIT_VIDEO) == -1)
    {
        // irjuk ki a hibat es termin�ljon a program
        std::cout << "[SDL initialization] Error during the SDL initialization: " << SDL_GetError() << std::endl;
        return;
    }

    SDL_GL_SetAttribute(SDL_GL_CONTEXT_PROFILE_MASK, SDL_GL_CONTEXT_PROFILE_CORE);

    SDL_GL_SetAttribute(SDL_GL_BUFFER_SIZE, 32);
    SDL_GL_SetAttribute(SDL_GL_RED_SIZE, 8);
    SDL_GL_SetAttribute(SDL_GL_GREEN_SIZE, 8);
    SDL_GL_SetAttribute(SDL_GL_BLUE_SIZE, 8);
    SDL_GL_SetAttribute(SDL_GL_ALPHA_SIZE, 8);
    // duplapufferel�s
    SDL_GL_SetAttribute(SDL_GL_DOUBLEBUFFER, 1);
    // m�lys�gi puffer h�ny bites legyen
    SDL_GL_SetAttribute(SDL_GL_DEPTH_SIZE, 24);

    win = 0;
    win = SDL_CreateWindow("Our City",		// az ablak fejl�ce
        50,						// az ablak bal-fels� sark�nak kezdeti X koordin�t�ja
        50,						// az ablak bal-fels� sark�nak kezdeti Y koordin�t�ja
        600,						// ablak sz�less�ge
        600,						// �s magass�ga
        SDL_WINDOW_OPENGL | SDL_WINDOW_SHOWN | SDL_WINDOW_RESIZABLE | SDL_WINDOW_ALLOW_HIGHDPI);

    if (win == 0)
    {
        std::cout << "[Ablak l�trehoz�sa]Hiba az SDL inicializ�l�sa k�zben: " << SDL_GetError() << std::endl;
        return;
    }

    context = SDL_GL_CreateContext(win);
    if (context == 0)
    {
        std::cout << "[OGL context creation] Error during the creation of the OGL context: " << SDL_GetError() << std::endl;
        return;
    }

    // 0 - nincs vsync
    // 1 - van vsync
    SDL_GL_SetSwapInterval(1);

    GLenum error = glewInit();
    if (error != GLEW_OK)
    {
        std::cout << "[GLEW] Error during the initialization of glew." << std::endl;
        return;
    }

    int glVersion[2] = { -1, -1 };
    glGetIntegerv(GL_MAJOR_VERSION, &glVersion[0]);
    glGetIntegerv(GL_MINOR_VERSION, &glVersion[1]);
    std::cout << "Running OpenGL " << glVersion[0] << "." << glVersion[1] << std::endl;

    if (glVersion[0] == -1 && glVersion[1] == -1)
    {
        SDL_GL_DeleteContext(context);
        SDL_DestroyWindow(win);

        std::cout << "[OGL context creation] Error during the inialization of the OGL context! Maybe one of the SDL_GL_SetAttribute(...) calls is erroneous." << std::endl;
        return;
    }

    // enged�lyezz�k �s �ll�tsuk be a debug callback f�ggv�nyt ha debug context-ben vagyunk 
    GLint context_flags;
    glGetIntegerv(GL_CONTEXT_FLAGS, &context_flags);
}

// ablak input eventek
void HandleEvents(SDL_Event ev, bool& quit)
{
    switch (ev.type)
    {
    case SDL_QUIT:
        std::cout << "quit" << std::endl;
        quit = true;
        break;
    case SDL_KEYDOWN:
        //if (ev.key.keysym.sym == SDLK_ESCAPE)
            //quit = true;
        //app->KeyboardDown(ev.key);
        break;
    case SDL_KEYUP:
        //app->KeyboardUp(ev.key);
        break;
    case SDL_MOUSEBUTTONDOWN:
        if (ev.button.button == SDL_BUTTON_LEFT)
        {
            mousePressed = true;
        }
        break;
    case SDL_MOUSEBUTTONUP:
        if (ev.button.button == SDL_BUTTON_LEFT)
        {
            mousePressed = false;
        }
        break;
    case SDL_MOUSEWHEEL:
        if (ev.wheel.y > 0) // scroll up
        {
            cameraDistance -= 0.2f * cameraDistance;
        }
        else if (ev.wheel.y < 0) // scroll down
        {
            cameraDistance += 0.2f * cameraDistance;
        }
        RotateCamera(0, 0);
        break;
    case SDL_MOUSEMOTION:
    {

        int x;
        int y;
        SDL_GetMouseState(&x, &y);

        if (mousePressed)
        {
            int dx = lastMouseX - x;
            int dy = lastMouseY - y;

            RotateCamera(dx, dy);
        }
        lastMouseX = x;
        lastMouseY = y;

        break;
    }

    case SDL_WINDOWEVENT:
        if (ev.window.event == SDL_WINDOWEVENT_SHOWN)
        {
            int w, h;
            SDL_GetWindowSize(win, &w, &h);

        }
        if (ev.window.event == SDL_WINDOWEVENT_SIZE_CHANGED)
        {

        }
        break;
    }
}

GLuint vboID;
GLuint vaoID;

GLuint vboPingPong[2];
GLuint vaoPingPong[2];

hipGraphicsResource* cudaVboResource[2];

Particle* particlesDev[2];
size_t particlesSize = numParticles * sizeof(Particle);
size_t size;
void InitVaoVbo()
{
    cudaGLSetGLDevice(0);

    //vao 1
    glGenVertexArrays(1, &vaoPingPong[0]);
    glBindVertexArray(vaoPingPong[0]);

    //vbo 1
    glGenBuffers(1, &vboPingPong[0]);
    glBindBuffer(GL_ARRAY_BUFFER, vboPingPong[0]);
    glBufferData(GL_ARRAY_BUFFER, numParticles * sizeof(Particle), nullptr, GL_DYNAMIC_DRAW);
    hipGraphicsGLRegisterBuffer(&cudaVboResource[0], vboPingPong[0], cudaGraphicsMapFlagsNone);

    glEnableVertexAttribArray(0);
    //pos
    glVertexAttribPointer((GLuint)0, 3, GL_FLOAT, GL_FALSE, sizeof(Particle), 0);
    //acc
    glEnableVertexAttribArray(1);
    glVertexAttribPointer((GLuint)1, 3, GL_FLOAT, GL_FALSE, sizeof(Particle), (void*)(sizeof(glm::vec3)));
    //mass
    glEnableVertexAttribArray(2);
    glVertexAttribPointer((GLuint)2, 1, GL_FLOAT, GL_FALSE, sizeof(Particle), (void*)(sizeof(glm::vec3) * 2));

    glBindVertexArray(0);
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    //vao 2
    glGenVertexArrays(1, &vaoPingPong[1]);
    glBindVertexArray(vaoPingPong[1]);

    //vbo 2
    glGenBuffers(1, &vboPingPong[1]);
    glBindBuffer(GL_ARRAY_BUFFER, vboPingPong[1]);
    glBufferData(GL_ARRAY_BUFFER, numParticles * sizeof(Particle), nullptr, GL_DYNAMIC_DRAW);
    hipGraphicsGLRegisterBuffer(&cudaVboResource[1], vboPingPong[1], cudaGraphicsMapFlagsNone);

    glEnableVertexAttribArray(0);
    //pos
    glVertexAttribPointer((GLuint)0, 3, GL_FLOAT, GL_FALSE, sizeof(Particle), 0);
    //acc
    glEnableVertexAttribArray(1);
    glVertexAttribPointer((GLuint)1, 3, GL_FLOAT, GL_FALSE, sizeof(Particle), (void*)(sizeof(glm::vec3)));
    //mass
    glEnableVertexAttribArray(2);
    glVertexAttribPointer((GLuint)2, 1, GL_FLOAT, GL_FALSE, sizeof(Particle), (void*)(sizeof(glm::vec3) * 2));

    glBindVertexArray(0);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
}

GLuint shaderProgramID;
void InitShaders()
{
    GLuint vs_ID = loadShader(GL_VERTEX_SHADER, "vert.vert");
    GLuint fs_ID = loadShader(GL_FRAGMENT_SHADER, "frag.frag");

    // a shadereket t�rol� program l�trehoz�sa
    shaderProgramID = glCreateProgram();

    // adjuk hozz� a programhoz a shadereket
    glAttachShader(shaderProgramID, vs_ID);
    glAttachShader(shaderProgramID, fs_ID);

    // attributomok osszerendelese a VAO es shader kozt
    glBindAttribLocation(shaderProgramID, 0, "vs_in_pos");
    glBindAttribLocation(shaderProgramID, 1, "vs_in_acc");
    glBindAttribLocation(shaderProgramID, 2, "vs_in_mass");

    // illessz�k �ssze a shadereket (kimen�-bemen� v�ltoz�k �sszerendel�se stb.)
    glLinkProgram(shaderProgramID);

    // linkel�s ellen�rzese
    GLint infoLogLength = 0, result = 0;

    glGetProgramiv(shaderProgramID, GL_LINK_STATUS, &result);
    glGetProgramiv(shaderProgramID, GL_INFO_LOG_LENGTH, &infoLogLength);
    if (GL_FALSE == result || infoLogLength != 0)
    {
        std::vector<char> VertexShaderErrorMessage(infoLogLength);
        glGetProgramInfoLog(shaderProgramID, infoLogLength, nullptr, VertexShaderErrorMessage.data());

        std::cerr << "[glLinkProgram] Shader linking error:\n" << &VertexShaderErrorMessage[0] << std::endl;
    }

    // m�r nincs ezekre sz�ks�g
    glDeleteShader(vs_ID);
    glDeleteShader(fs_ID);
}

void InitParticles()
{
    for (int i = 0; i < numParticles; i++) {
        // Generate random angle and distance from the center
        float angle = ((float)rand() / RAND_MAX) * 2 * M_PI;
        float distance = ((float)rand() / RAND_MAX) * 0.5f;

        // Calculate initial positions around the center
        particlesHost[i].x = 0.0f + cos(angle) * distance;
        particlesHost[i].y = 0.0f + sin(angle) * distance;
        particlesHost[i].z = (-0.5f + ((float)rand() / RAND_MAX)) * 0.1f;

        // Calculate initial velocities for rotation
        if (initialVel)
        {
            float speed = distance * distance * 100.0f;
            particlesHost[i].vx = -sin(angle) * speed;
            particlesHost[i].vy = cos(angle) * speed;
            particlesHost[i].vz = 0.0f;
        }
        else
        {
            particlesHost[i].vx = 0.0f;
            particlesHost[i].vy = 0.0f;
            particlesHost[i].vz = 0.0f;
        }
        

        particlesHost[i].mass = ((float)rand() / RAND_MAX);
    }
}

void InitRendering()
{
    glClearColor(0.125f, 0.25f, 0.5f, 1.0f);

    glEnable(GL_CULL_FACE);
    glEnable(GL_DEPTH_TEST);
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
    glPointSize(2.0);

    //InitVaoVbo();
    InitShaders();
}

void RenderParticles(SDL_Window* window, glm::mat4 mvp, int bufferID)
{
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // shader bekapcsol�sa, ebben a projektben a teljes programot jel�li, hisz nem v�ltunk a shaderek k�z�tt
    glUseProgram(shaderProgramID);

    // kapcsoljuk be a VAO-t (a VBO j�n vele egy�tt)
    glBindVertexArray(vaoPingPong[bufferID]);

    // uniformok
    glm::mat4 locMVP = mvp;
    glUniformMatrix4fv(glGetUniformLocation(shaderProgramID, "MVP"), 1, GL_FALSE, &locMVP[0][0]);

    glDrawArrays(GL_POINTS, 0, numParticles);

    // VAO kikapcsol�sa
    glBindVertexArray(0);

    // shader kikapcsol�sa
    glUseProgram(0);

    //SDL_GL_SwapWindow(win);
}
void Restart()
{
    hipGraphicsUnregisterResource(cudaVboResource[0]);
    hipGraphicsUnregisterResource(cudaVboResource[1]);
    glDeleteBuffers(1, &vboPingPong[0]);
    glDeleteBuffers(1, &vboPingPong[1]);

    free(particlesHost);

    grav = gravSetting;
    numParticles = (int)startingParticleNum;
    particlesHost = (Particle*)malloc(numParticles * sizeof(Particle));
    InitParticles();

    InitVaoVbo();
    particlesSize = numParticles * sizeof(Particle);

    // map cuda
    // get pointer
    // copy data
    // unmap cuda
    //size_t size;
    hipGraphicsMapResources(1, &cudaVboResource[0]);
    hipGraphicsResourceGetMappedPointer((void**)&particlesDev[0], &size, cudaVboResource[0]);
    hipMemcpy(particlesDev[0], particlesHost, particlesSize, hipMemcpyHostToDevice);
    hipGraphicsUnmapResources(1, &cudaVboResource[0]);

    hipGraphicsMapResources(1, &cudaVboResource[1]);
    hipGraphicsResourceGetMappedPointer((void**)&particlesDev[1], &size, cudaVboResource[1]);
    hipMemcpy(particlesDev[1], particlesHost, particlesSize, hipMemcpyHostToDevice);
    hipGraphicsUnmapResources(1, &cudaVboResource[1]);

}
void ShowMenuUI()
{
    if (ImGui::Begin("Gyak11"))
    {
        ImGui::Text("Gravitational constant");
        ImGui::InputFloat(" ", &gravSetting, 0.1f, 0.1f, 1);

        ImGui::Text("Particle number");
        ImGui::InputFloat("  ", &startingParticleNum, 100.0f, 100.0f, 0);

        if (ImGui::Checkbox("Uniform distribution", &uniform))
        {

        }

        if (ImGui::Checkbox("Initial velocity", &initialVel))
        {

        }

        if (ImGui::Button("Restart"))
        {
            Restart();
        }
    }
    ImGui::End();
}
int main(int argc, char* argv[]) {
    bool quit = false;
    //InitParticles();

    InitSDLWindow();
    Restart();
    glm::mat4 projection = glm::perspective(glm::radians(45.0f), 1.0f, 0.1f, 100.0f);
    RotateCamera(0, 0);

    InitRendering();
    //Imgui init
    ImGui_ImplSdlGL3_Init(win);

    // Start the simulation loop
    for (int i = 0; i < numIterations; i++) {
        while (SDL_PollEvent(&event))
        {
            ImGui_ImplSdlGL3_ProcessEvent(&event);
            HandleEvents(event, quit);
        }
        if (quit)
        {
            break;
        }

        ImGui_ImplSdlGL3_NewFrame(win);

        // Swap particle buffers
        int currentBuffer = i % 2;
        int nextBuffer = (i + 1) % 2;

        hipGraphicsMapResources(1, &cudaVboResource[0]);
        hipGraphicsResourceGetMappedPointer((void**)&particlesDev[0], &size, cudaVboResource[0]);
        
        hipGraphicsMapResources(1, &cudaVboResource[1]);
        hipGraphicsResourceGetMappedPointer((void**)&particlesDev[1], &size, cudaVboResource[1]);
        
        // Launch the kernel
        nBodySimulation <<< (numParticles + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (particlesDev[currentBuffer], particlesDev[nextBuffer], dt, numParticles, grav);

        // Wait for kernel to finish
        hipDeviceSynchronize();
        hipGraphicsUnmapResources(1, &cudaVboResource[0]);
        hipGraphicsUnmapResources(1, &cudaVboResource[1]);

        RenderParticles(win, projection * view, currentBuffer);

        
        ShowMenuUI();

        ImGui::Render();
        SDL_GL_SwapWindow(win);

    }

    // Cleanup
    ImGui_ImplSdlGL3_Shutdown();
    hipGraphicsUnregisterResource(cudaVboResource[0]);
    hipGraphicsUnregisterResource(cudaVboResource[1]);
    glDeleteBuffers(1, &vboPingPong[0]);
    glDeleteBuffers(1, &vboPingPong[1]);

    free(particlesHost);

    SDL_GL_DeleteContext(context);
    SDL_DestroyWindow(win);

    return 0;
}